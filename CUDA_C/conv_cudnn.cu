#include "hip/hip_runtime.h"
// Includes
#include <stdio.h>
#include "timer.h"
#include <hipDNN.h>

// Defines
#define epsilon (float)1e-4
#define verbose 0

#define C 3
#define H 1024
#define W 1024
#define K 64
#define FH 3
#define FW 3
#define P 1


// Create input matrix stored in host memory.
double* createIMatrix() {
  double* input_matrix;
  input_matrix = (double*) malloc(C * H * W * sizeof(double));
  for(int c = 0; c < C; c++) {
    for(int h = 0; h < H; h++) {
        for(int w = 0; w < W; w++) {
            input_matrix[c * H * W + h * W + w] = c * (h + w);
        }
    }
  }
  return input_matrix;
}

// Create all K filter matrices on host
double* createFilterMatrices() {
  double* filters;
  filters = (double*) malloc(K * C * FH * FW * sizeof(double));
  for(int k = 0; k < K; k++) {
    for(int c = 0; c < C; c++) {
        for(int h = 0; h < FH; h++) {
            for(int w = 0; w < FW; w++) {
                filters[k * C * FH * FW + c * FH * FW + h * FW + w] = (c + k) * (h + w);
            }
        }
    }
  }
  return filters;
}

// Compute and check the checksum of the result matrix
// expected to be 122756344698240
void checkResult(double* result) {

  double checksum = 122756344698240;
  double checksum_M = 0;
  
  for(int k = 0; k < K; k++) {
    for(int h = 0; h < H; h++) {
        for(int w = 0; w < W; w++) {
            checksum_M += result[k * H * W + h * W + w];
        }
    }
  }
  printf("checksum: %lf\n", checksum_M);
  if(fabs(checksum - checksum_M)> epsilon * checksum) {
    printf("\n\nTEST FAILED\n");
    printf("computed checksum: %lf\n", checksum_M);
    printf("actual checksum: %lf\n", checksum);
  }
}

//
// main
//
int main(int argc, char** argv) {
  hipdnnHandle_t cudnn;
  hipdnnCreate(&cudnn);

  int image_bytes = C * H * W * sizeof(double);
  double* input_matrix = createIMatrix();
  int filters_bytes = K * C * FH * FW * sizeof(double);
  double* filters = createFilterMatrices();
  int result_bytes = K * H * W * sizeof(double);
  double* result;
  result = (double*) malloc(result_bytes);

  // descriptors
  hipdnnTensorDescriptor_t input_descriptor;
  hipdnnFilterDescriptor_t kernel_descriptor;
  hipdnnTensorDescriptor_t output_descriptor;
  hipdnnConvolutionDescriptor_t convolution_descriptor;

  hipdnnCreateTensorDescriptor(&input_descriptor);
  hipdnnCreateFilterDescriptor(&kernel_descriptor);
  hipdnnCreateTensorDescriptor(&output_descriptor);
  hipdnnCreateConvolutionDescriptor(&convolution_descriptor);

  hipdnnSetTensor4dDescriptor(input_descriptor,
    /*format=*/HIPDNN_TENSOR_NCHW,
    /*dataType=*/HIPDNN_DATA_DOUBLE,
    /*batch_size=*/1,
    /*channels=*/C,
    /*image_height=*/H,
    /*image_width=*/W);

  hipdnnSetFilter4dDescriptor(kernel_descriptor,
    /*dataType=*/HIPDNN_DATA_DOUBLE,
    /*format=*/HIPDNN_TENSOR_NCHW,
    /*out_channels=*/K,
    /*in_channels=*/C,
    /*kernel_height=*/FH,
    /*kernel_width=*/FW);

  hipdnnSetConvolution2dDescriptor(convolution_descriptor,
    /*pad_height=*/P,
    /*pad_width=*/P,
    /*vertical_stride=*/1,
    /*horizontal_stride=*/1,
    /*dilation_height=*/1,
    /*dilation_width=*/1,
    /*mode=*/HIPDNN_CONVOLUTION,
    /*computeType=*/HIPDNN_DATA_DOUBLE);
  
  hipdnnSetTensor4dDescriptor(output_descriptor,
    /*format=*/HIPDNN_TENSOR_NCHW,
    /*dataType=*/HIPDNN_DATA_DOUBLE,
    /*batch_size=*/1,
    /*channels=*/K,
    /*image_height=*/H,
    /*image_width=*/W);

  double* d_input;
  hipMallocManaged(&d_input, image_bytes);
  hipMemcpy(d_input, input_matrix, image_bytes, hipMemcpyHostToDevice);
  
  double* d_kernel;
  hipMallocManaged(&d_kernel, filters_bytes);
  hipMemcpy(d_kernel, filters, filters_bytes, hipMemcpyHostToDevice);

  double* d_output;
  hipMallocManaged(&d_output, result_bytes);

  const int n_requestedAlgo = 10;
  int n_returnedAlgo;
  hipdnnConvolutionFwdAlgoPerf_t fwd_algo_perf[n_requestedAlgo];
  hipdnnFindConvolutionForwardAlgorithm(cudnn, input_descriptor, kernel_descriptor, convolution_descriptor, output_descriptor, n_requestedAlgo, &n_returnedAlgo, fwd_algo_perf);

  hipdnnConvolutionFwdAlgo_t convolution_algorithm;
  convolution_algorithm = fwd_algo_perf[0].algo;

  size_t workspace_bytes = 0;
  hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
    input_descriptor,
    kernel_descriptor,
    convolution_descriptor,
    output_descriptor,
    convolution_algorithm,
    &workspace_bytes);

  double* d_workspace;
  hipMalloc(&d_workspace, workspace_bytes);

  double alpha = 1, beta = 0;

    // warmup
  hipdnnConvolutionForward(cudnn,
    &alpha,
    input_descriptor,
    d_input,
    kernel_descriptor,
    d_kernel,
    convolution_descriptor,
    convolution_algorithm,
    d_workspace,
    workspace_bytes,
    &beta,
    output_descriptor,
    d_output);

  hipDeviceSynchronize();
  
    // Set up timer
  initialize_timer();
  start_timer();

  hipdnnConvolutionForward(cudnn,
    &alpha,
    input_descriptor,
    d_input,
    kernel_descriptor,
    d_kernel,
    convolution_descriptor,
    convolution_algorithm,
    d_workspace,
    workspace_bytes,
    &beta,
    output_descriptor,
    d_output);

  hipDeviceSynchronize();

  stop_timer();

  hipMemcpy(result, d_output, result_bytes, hipMemcpyDeviceToHost);

  double time = elapsed_time();
  printf( "Time: %lf (sec)\n", time);

  // Verify that the result is correct.
  checkResult(result);

  hipFree(d_kernel);
  hipFree(d_input);
  hipFree(d_output);
  hipFree(d_workspace);

  hipdnnDestroyTensorDescriptor(input_descriptor);
  hipdnnDestroyTensorDescriptor(output_descriptor);
  hipdnnDestroyFilterDescriptor(kernel_descriptor);
  hipdnnDestroyConvolutionDescriptor(convolution_descriptor);

  hipdnnDestroy(cudnn);

  free(input_matrix);
  free(filters);
  free(result);
}
